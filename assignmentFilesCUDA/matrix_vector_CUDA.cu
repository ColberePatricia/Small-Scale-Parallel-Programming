#include <stdlib.h>
#include <stdio.h>
#include <iostream>
#include <hip/hip_runtime.h>  // For CUDA runtime API
#include <hip/hip_runtime_api.h>  // For checkCudaError macro
#include <helper_timer.h>  // For CUDA SDK timers
#include "wtime.h"
#include "mmio.h"
#include "matrixPreprocessing.h"
#include "test.h"
#include "matrixVector.h"

//Simple dimension: define a 1D block structure
#define BD 256
const dim3 BLOCK_DIM(BD);

// Generates a random vector
void generateVector(int vectorSize, double* vector) {
	srand(12345);
	for (int row = 0; row < vectorSize; ++row) {
		vector[row] = 100.0f * ((double)rand()) / RAND_MAX;
	}
}

// Generates a random matrix
void generateMatrix(int rows, int cols, double* matrix) {
	srand(21345);
	int idx;
	for (int row = 0; row < rows; ++row) {
		for (int col = 0; col < cols; ++col) {
			idx = row * cols + col;
			matrix[idx] = 100.0f * ((double)rand()) / RAND_MAX;
		}
	}
}




int main(int argc, char** argv)
{

	testMatrixProcessing();
	testMatrixVectorProduct();

	char* fileName;
	if (argc < 2)
	{
		fprintf(stderr, "Usage: %s [martix-market-filename]\nWe will be using the cage4 matrix as default\n", argv[0]);
		//fileName = "D:\\Cranfield work\\Small Scale Parallel Programming\\matrices\\cage4.mtx";
		fileName = "../matrices/cage4.mtx";
	}
	else {
		fileName = argv[1];
	}


	// We read the file of the matrix
	int ret_code;
	MM_typecode matcode;
	FILE *f;
	int M, N, nz;
	int i, *I, *J;
	double *val;

	// If the file of the matrix cannot be opened
	if ((f = fopen(fileName, "r")) == NULL)
		fprintf(stdout, "The file %s could not be opened\n", fileName);
	else if (mm_read_banner(f, &matcode) != 0)
		fprintf(stdout, "Could not process Matrix Market banner.\n");
	else if (mm_is_complex(matcode) && mm_is_matrix(matcode) && mm_is_sparse(matcode))
		fprintf(stdout, "Sorry, this application does not support Market Market type: [%s]\n", mm_typecode_to_str(matcode));
	else if ((ret_code = mm_read_mtx_crd_size(f, &M, &N, &nz)) != 0)
		fprintf(stdout, "Could not read the size of the matrix");
	else {
		/* reseve memory for matrices */
		I = (int *)malloc(nz * sizeof(int));
		J = (int *)malloc(nz * sizeof(int));
		val = (double *)malloc(nz * sizeof(double));
		/* NOTE: when reading in doubles, ANSI C requires the use of the "l"  */
		/*   specifier as in "%lg", "%lf", "%le", otherwise errors will occur */
		/*  (ANSI C X3.159-1989, Sec. 4.9.6.2, p. 136 lines 13-15)            */
		for (i = 0; i < nz; i++) {
			fscanf(f, "%d %d %lg\n", &I[i], &J[i], &val[i]);
			I[i]--;  // adjust from 1-based to 0-based
			J[i]--;
		}
		if (f != stdin) fclose(f);


		// We now have the matrix with I, J and val
	// We will generate our CSR matrix from them
		int* IRP = (int *)malloc((M + 1) * sizeof(int));
		int* JA_CSR = (int *)malloc(nz * sizeof(int));
		double* AS_CSR = (double *)malloc(nz * sizeof(double));
		IRP = getCSR_IRP(M, nz, I);
		JA_CSR = getCSR_JA(nz, I, J);
		AS_CSR = getCSR_AS(nz, I, val);

		// We will generate our ELLPACK matrix
		int MAXNZ;
		MAXNZ = getELLPACK_MAXNZ(nz, I);
		int* JA_ELLPACK = (int *)malloc(M * MAXNZ * sizeof(int));
		double* AS_ELLPACK = (double *)malloc(M * MAXNZ * sizeof(double));
		JA_ELLPACK = getELLPACK_JA(M, nz, I, J, MAXNZ);
		AS_ELLPACK = getELLPACK_AS(M, nz, I, val, MAXNZ);


		// We initiate our matrices for the product
		double* x = (double*)malloc(sizeof(double)*N);
		double* y = (double*)malloc(sizeof(double)*M);

		// We generate randomly x of size N
		generateVector(N, x);



		// We create our CUDA matrices
		double *d_AS_CSR, *d_AS_ELLPACK, *d_x, *d_y;
		int *d_IRP, *d_JA_CSR, *d_JA_ELLPACK;
		checkCudaErrors(hipMalloc((void**)&d_AS_CSR, nz * sizeof(double)));
		checkCudaErrors(hipMalloc((void**)&d_AS_ELLPACK, M * MAXNZ * sizeof(double)));
		checkCudaErrors(hipMalloc((void**)&d_x, N * sizeof(double)));
		checkCudaErrors(hipMalloc((void**)&d_y, M * sizeof(double)));
		checkCudaErrors(hipMalloc((void**)&d_IRP, (M + 1) * sizeof(int)));
		checkCudaErrors(hipMalloc((void**)&d_JA_CSR, nz * sizeof(int)));
		checkCudaErrors(hipMalloc((void**)&d_JA_ELLPACK, M * MAXNZ * sizeof(int)));

		// Copy matrices from the host (CPU) to the device (GPU).
		checkCudaErrors(hipMemcpy(d_AS_CSR, AS_CSR, nz * sizeof(double), hipMemcpyHostToDevice));
		checkCudaErrors(hipMemcpy(d_AS_ELLPACK, AS_ELLPACK, M * MAXNZ * sizeof(double), hipMemcpyHostToDevice));
		checkCudaErrors(hipMemcpy(d_x, x, N * sizeof(double), hipMemcpyHostToDevice));
		checkCudaErrors(hipMemcpy(d_IRP, IRP, (M + 1) * sizeof(int), hipMemcpyHostToDevice));
		checkCudaErrors(hipMemcpy(d_JA_CSR, JA_CSR, nz * sizeof(int), hipMemcpyHostToDevice));
		checkCudaErrors(hipMemcpy(d_JA_ELLPACK, JA_ELLPACK, M * MAXNZ * sizeof(int), hipMemcpyHostToDevice));


		// Calculate the dimension of the grid of blocks (1D) necessary to cover all rows.
		const dim3 GRID_DIM((M - 1 + BLOCK_DIM.x) / BLOCK_DIM.x, 1);
		double flopcnt = 2.e-6*M*N;

		// Create the CUDA SDK timer.
		StopWatchInterface* timer = 0;
		sdkCreateTimer(&timer);


		// We do the product with CSR
		timer->start();
		for (int i=0;i<10;i++)
			MatrixVectorCSR << <GRID_DIM, BLOCK_DIM >> > (M, d_IRP, d_JA_CSR, d_AS_CSR, d_x, d_y);
		checkCudaErrors(hipDeviceSynchronize());
		timer->stop();
		double gpuflops = 10 * flopcnt / timer->getTime();

		// We print our results
		fprintf(stdout, "CSR: Matrix-Vector product of size %d x %d: time %lf  GFLOPS %lf \n", M, N, (timer->getTime())/10, gpuflops);


		// We do the product for ELLPACK
		timer->reset();
		timer->start();
		for (int i = 0;i < 10;i++)
			MatrixVectorELLPACK << <GRID_DIM, BLOCK_DIM >> > (M, N, MAXNZ, d_JA_ELLPACK, d_AS_ELLPACK, d_x, d_y);
		checkCudaErrors(hipDeviceSynchronize());
		timer->stop();
		gpuflops = 10 * flopcnt / timer->getTime();

		// We print our results
		fprintf(stdout, "ELLPACK: Matrix-Vector product of size %d x %d: time %lf  GFLOPS %lf \n", M, N, (timer->getTime())/10, gpuflops);



		// We free the matrices and vectors

		delete timer;

		checkCudaErrors(hipFree(d_IRP));
		checkCudaErrors(hipFree(d_x));
		checkCudaErrors(hipFree(d_JA_CSR));
		checkCudaErrors(hipFree(d_AS_CSR));
		checkCudaErrors(hipFree(d_JA_ELLPACK));
		checkCudaErrors(hipFree(d_AS_ELLPACK));

		free(IRP);
		free(JA_CSR);
		free(AS_CSR);
		free(JA_ELLPACK);
		free(AS_ELLPACK);
		free(x);
		free(y);
	}

  return 0;
}
