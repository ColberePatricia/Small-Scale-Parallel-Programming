#include "hip/hip_runtime.h"
#include "matrixVector.h"

// Simple CPU implementation of matrix-vector product
double* MatrixVector(int rows, int cols, const double* A, const double* x, double* y) {
	int row, col, idx;
	double t;
	for (row = 0; row < rows; ++row) {
		t = 0.0;
		for (col = 0; col < cols; ++col) {
			idx = row * cols + col;
			t = t + A[idx] * x[col];
		}
		y[row] = t;
	}
	return y;
}

// Implementation of matrix-vector product with the matrix in CSR
// using one thread per row
__global__ void MatrixVectorCSR(int M, int* IRP, int* JA, double* AS, double* x, double* y) {
	double temp;
	int tr = threadIdx.x;
	int i = blockIdx.x*blockDim.x + tr;
	if (i < M) {
		temp = 0;
		for (int j = IRP[i];j <= IRP[i + 1] - 1;j++) {
			temp += AS[j] * x[JA[j]];
		}
		y[i] = temp;
	}
}




// Implementation of matrix-vector product with the matrix in ELLPACK
// using a block of threads for each block of rows.
__global__ void MatrixVectorELLPACK(int M, int N, int MAXNZ, int* JA, double* AS, double* x, double* y) {
	__shared__ double ax[16][64];
	double temp;
	int tr     = threadIdx.y;
  int tc     = threadIdx.x;
  int i    = blockIdx.x*blockDim.y + tr;
  ax[tr][tc] = 0.0;
	if (i < M) {
		int idx = i * MAXNZ + tc;
		temp = 0;
		int j;
		for (j = tc;j < MAXNZ;j+=64) {
			temp += AS[idx] * x[JA[idx]];
			idx+=64; // The size of JA is M * MAXNZ
		}
		if (j<MAXNZ){
			temp += AS[idx] * x[JA[idx]];
		}
		ax[tr][tc] = temp;
	}
	__syncthreads();
	for (int s=64/2; s >32; s >>=1){
		if (tc<s)
			ax[tr][tc] += ax[tr][tc+s];
			__syncthreads();
	}
	for (int s=min(32,64/2); s >0; s >>=1){
		if (tc<s)
			ax[tr][tc] += ax[tr][tc+s];
	}

	if ((tc == 0)&&(i<M))
		y[i] = ax[tr][tc];
}

